
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <string>
//
#include <hip/hip_complex.h>
//
#define MaxIteration 255;  //!< Le nombre max d'itération est 255, soit de base le blanc.
//
static void HandleError(	hipError_t err,
                            const char *file,
                            int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
// Cette méthode sert uniquement à sauvegarder le vecteur sous forme d'une image en niveau de gris sur 8 bits.
void save_pgm(  const char*                         filename,
                const size_t                        width,
                const size_t                        height,
                const std::vector<std::uint8_t>&   data)
{
    std::ofstream fout{ filename };
    // L'en-tête
    fout << "P2\n" << width << " " << height << " 255\n";
    for (size_t row = 0; row < height; ++row)
    {
        for (size_t col = 0; col < width; ++col)
        {
            fout << (col ? " " : "")
                 << static_cast<unsigned>(data[row * width + col]);
        }
        fout << "\n";
    }
    fout.close();
}
//
__global__ void mandel_kernel_double(int width, int height, uint8_t* ret)
{
    // TODO : garder le même algorithme qu'en C++, il faut juste "traduire" les éléments
    // de C++ en CUDA.
    // Que devient la boucle 'for' si on souhaite calculer chaque pixel dans un thread ?
    // Attention à l'indice global du thread, il faut vérifier que nous sommes bien dans les bornes de l'image !
    // Utiliser les méthodes et les éléments fournis dans le PDF concernant les nombres complexes avec CUDA 
    // (on peut aussi explorer 'cuComplex.h' pour voir ce qui s'y trame).

  double aspect = (double) width / height;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height)
  {
    double myrow = (double) y;
    double mycol = (double) x;
    myrow /= height;
    mycol /= width;
    hipDoubleComplex z0 = make_hipDoubleComplex(aspect * (2 * mycol - 1) - 0.5, 2 * myrow - 1);
    double mag = 0.0;
    hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
    uint8_t lim = MaxIteration;
    do
    {
      z = hipCadd(hipCmul(z, z), z0);
      mag = hipCabs(z);
    } while (lim-- && mag < 4.0);
    ret[y * width + x] = lim;
  }
}
//
int main(int argc, char* argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage:\n"
                  << argv[0] << " [width] [height]\n";
        return 1;
    }
    const size_t width  = std::stoul(argv[1]);
    const size_t height = std::stoul(argv[2]);
    std::vector<std::uint8_t> image(height * width, 0);
    // Note : il est possible de manipuler le pointeur de données sous-jacent au vecteur via la méthode '.data()'
    auto t0 = std::chrono::high_resolution_clock::now();
    // TODO : Appeler mandel_kernel_double
    uint8_t *d_image;
    HANDLE_ERROR(hipMalloc(&d_image, width * height * sizeof(uint8_t)));
    HANDLE_ERROR(hipMemcpy(d_image, image.data(), width * height * sizeof(uint8_t), hipMemcpyHostToDevice));
    int blockSize = 32;
    dim3 dimGrid((width + blockSize - 1) / blockSize, (height + blockSize - 1) / blockSize);

    mandel_kernel_double<<<dimGrid, dim3(blockSize, blockSize)>>>(width, height, d_image);
    HANDLE_ERROR(hipMemcpy(image.data(), d_image, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost));
    auto t1 = std::chrono::high_resolution_clock::now();
    std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << std::chrono::duration<double>(t1-t0).count() << " seconds (GPU version)\n";
    save_pgm("output_GPU.pgm", width, height, image);
}
