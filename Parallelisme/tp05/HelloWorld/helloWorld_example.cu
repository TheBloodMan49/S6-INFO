
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}
//
void c_hello()
{
    printf("Hello World from CPU in C!\n");
}
//
void cpp_hello()
{
    std::cout << "Hello World from CPU in C++!" << std::endl;
}
//
int main()
{
    // Call CUDA
    cuda_hello<<<1,1>>>(); 
    // Tester avec et sans cette ligne :
    hipDeviceSynchronize();
    
    // Call CPU - C version
    c_hello();
    
    // Call CPU - C++ version
    cpp_hello();
    return 0;
}