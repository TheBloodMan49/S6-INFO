#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <string>
#include <sstream>

static void HandleError(	hipError_t err,
  const char *file,
  int line )
{
  if (err != hipSuccess)
  {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
    file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// Cette méthode ouvre une image pmg et la stocke dans un vecteur 8 bits.
std::vector<std::uint8_t> open_pgm(const std::string & filename, int& numrows, int& numcols)
{
  int row = 0, col = 0, tmp;
  std::ifstream infile{filename};
  std::stringstream ss;
  std::string inputLine = "";

  std::cout << "Reading input image..." << std::endl;

  // First line : version
  getline(infile,inputLine);
  if(inputLine.compare("P2") != 0) std::cerr << "Version error" << std::endl;
  else std::cout << "| Version : " << inputLine << std::endl;

  // Continue with a stringstream
  ss << infile.rdbuf();
  // Third line : size
  ss >> numcols >> numrows >> tmp;
  std::cout << "| " << numcols << " columns and " << numrows << " rows" << std::endl;

  std::vector<std::uint8_t> data(numcols * numrows);

  // Following lines : data
  for(row = 0; row < numrows; ++row)
  {
    for (col = 0; col < numcols; ++col) 
    {
      ss >> tmp;
      data[row*numcols + col] = tmp;
    }
  }

  infile.close();

  return data;
}

// Cette méthode sauvegarder un vecteur sous forme d'une image en niveau de gris sur 8 bits.
void save_pgm(  const std::string&                 filename,
                const size_t                       width,
                const size_t                       height,
                const std::vector<std::uint8_t>&   data)
{
    std::ofstream fout{ filename };
    // Header
    fout << "P2\n" << width << " " << height << " 255\n";
    for (size_t row = 0; row < height; ++row)
    {
        for (size_t col = 0; col < width; ++col)
        {
            fout << (col ? " " : "")
                 << static_cast<unsigned>(data[row * width + col]);
        }
        fout << "\n";
    }
    fout.close();

    std::cout << "Save image done!" << std::endl;
}
//
__global__ void convol_kernel(/* TODO */)
{
 // TODO
}
//
std::vector<std::uint8_t> apply_convolution(
  const std::vector<std::uint8_t> & input_image, 
  int numrows, 
  int numcols, 
  int numiter)
{    
    // Allocate output image (same size) and usefull buffer
    dim3 block_size(16, 16);
    dim3 grid_size(numcols / block_size.x, numrows / block_size.y);
    // TODO
    
    // Copy data from RAM to VRAM
    // TODO
    
    // Call kernel
    for (int i = 0; i < numiter; i++)
    {
      convol_kernel<<<grid_size, block_size>>>(output_image_gpu, buffer_image_gpu, numrows, numcols);
      std::swap(output_image_gpu, buffer_image_gpu);
    }

    // Get resuts on RAM
    std::vector<std::uint8_t> output_image(input_image.size());
    HANDLE_ERROR(hipMemcpy(output_image.data(), output_image_gpu, sizeof(std::uint8_t) * input_image.size(), hipMemcpyDeviceToHost));

    std::cout << "Convolution ... done" << std::endl;
    return output_image;
}
//
int main(int argc, char* argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage:\n"
                  << argv[0] << " [PATH/image.pgm] [number of convolution iterations. ex:100]\n";
        return 1;
    }

    std::string input_file = argv[1];
    int num_iter = std::stoul(argv[2]);

    std::string output_file {"image_convolution_GPU.pgm"};

    int numrows, numcols;

    // Read input image
    const std::vector<std::uint8_t>& input_image = open_pgm(input_file, numrows, numcols);

    // Do convolution
    auto t0 = std::chrono::high_resolution_clock::now();
    const std::vector<std::uint8_t>& output_image = apply_convolution(input_image, numrows, numcols, num_iter);
    auto t1 = std::chrono::high_resolution_clock::now();
    std::cout << "Image convolution done in " << std::chrono::duration<double>(t1-t0).count() << " seconds (GPU version)\n";    

    // Export output image
    save_pgm(output_file, numcols, numrows, output_image);
}
