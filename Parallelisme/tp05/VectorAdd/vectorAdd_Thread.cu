#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <time.h>
//
static void HandleError(    hipError_t err,
							const char *file,
							int line )
{
	if (err != hipSuccess)
	{
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
		file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//
#define N 10000000U
#define epsilon 1E-6f
//
void vector_add_c(  float*  pOut,
					float*  pIn_1,
					float*  pIn_2,
					int     n)
{
	for(int i = 0; i < n; i++)
	{
		pOut[i] = pIn_1[i] + pIn_2[i];
	}
}
// Le mot clé '__global__' indique que cette méthode est appelée par le CPU ('host') et exécutée sur le GPU ('device').
__global__ void vector_add_basic_cuda(  float*  dOut,
										float*  dIn_1,
										float*  dIn_2,
										int     n)
{
	for(int i = 0; i < n; i++)
	{
		dOut[i] = dIn_1[i] + dIn_2[i];
	}
}

__global__ void vector_add_thread_cuda(	float*  dOut,
										float*  dIn_1,
										float*  dIn_2,
										int	 n)
{
  for (int i = threadIdx.x; i < n; i += blockDim.x)
  {
    dOut[i] = dIn_1[i] + dIn_2[i];
  }
}

__global__ void vector_add_grid_cuda(	float*  dOut,
	float*  dIn_1,
	float*  dIn_2,
	int	 n)
{
	  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
  {
    dOut[i] = dIn_1[i] + dIn_2[i];
  }
}

//
int main()
{
  try{
	// Tableau CPU
	float *pOut     = NULL;
	float *pIn_1    = NULL;
	float *pIn_2    = NULL;
	
	// Tableau GPU : à ce stade, une adresse mémoire
	float *dOut     = NULL;
	float *dIn_1    = NULL;
	float *dIn_2    = NULL;

	//  Allocation de la mémoire
	pOut    = (float*)malloc(sizeof(float) * N);
	pIn_1   = (float*)malloc(sizeof(float) * N);
	pIn_2   = (float*)malloc(sizeof(float) * N);

	// Allocation de la mémoire sur le GPU
	// cudaMalloc(void **devPtr, size_t count) : Permet d'allouer 'count' bits sur le GPU.
	HANDLE_ERROR(hipMalloc((void**)&dOut,  sizeof(float) * N));
	HANDLE_ERROR(hipMalloc((void**)&dIn_1, sizeof(float) * N));
	HANDLE_ERROR(hipMalloc((void**)&dIn_2, sizeof(float) * N));

	// Initialisation des tableaux.
	for(int i = 0; i < N; i++)
	{
		pIn_1[i] = 1.0f; 
		pIn_2[i] = 2.0f;
		// Les lignes suivants compileraient mais provoqueraient un crash à l'exécution :
		// dIn_1[i] = 1.0f; 
		// dIn_2[i] = 2.0f;
		// Il n'est en effet pas possible d'accéder à de la mémoire dite 'device', c'est à dire allouée
		// via 'cudaMalloc' directement depuis le CPU...
		// Il faut faire des copies !
	}

	// On va maintenant copier ces tableaux sur le GPU, plus précisément :
	// copier le contenu de 'pIn_1' dans 'dIn_1'
	// copier le contenu de 'pIn_2' dans 'dIn_2'
	// cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind) : copie les 'count' bits de 'src' vers 'dst'
	HANDLE_ERROR(hipMemcpy(dIn_1, pIn_1, sizeof(float) * N, hipMemcpyHostToDevice));
	// Si l'ordre de transfert des données n'est pas correct, une erreur est remontée :
	HANDLE_ERROR(hipMemcpy(dIn_2, pIn_2, sizeof(float) * N, hipMemcpyHostToDevice));

	// Addition des deux vecteurs.
	vector_add_c(pOut, pIn_1, pIn_2, N);

  // Time
  int start = clock();
  vector_add_thread_cuda<<<1, 256>>>(dOut, dIn_1, dIn_2, N);
  hipDeviceSynchronize();
  int end = clock();
  std::cout << "Time : " << (end - start) / (double) CLOCKS_PER_SEC << std::endl;

	// La ligne suivante compile mais l'exécution crash.
	// En effet de la même façon qu'il n'est pas possible d'accéder directement à de la mémoire GPU ('device') depuis le CPU ('host'),
	// l'inverse n'est pas non plus possible.
	//vector_add_basic_cuda<<<1, 1>>>(dOut, pIn_1, pIn_2, N);

    // On copie cette fois-ci les données de 'dOut' sur le CPU dans 'pOut_fromGPU'.
    float *pOut_fromGPU     = NULL;
    pOut_fromGPU            = (float*)malloc(sizeof(float) * N);
	// Cette fois-ci, il faut transférer des données depuis le GPU ('device') vers le CPU ('host').
    HANDLE_ERROR(hipMemcpy(pOut_fromGPU, dOut, sizeof(float) * N, hipMemcpyDeviceToHost));
	// On affiche la première sortie du résultat :
	std::cout << "First value in CPU buffer : " << pOut[0] << std::endl;

	float first_value_on_gpu = pOut_fromGPU[0];
	std::cout << "First value in GPU buffer : " << first_value_on_gpu << std::endl;
	// Comment copier la première valeur contenu dans le buffer 'dOut' dans un float ?

	// Vérification que le résultat est correct sur le CPU.
	int i;
	for(i = 0; i < N; i++)
	{
		if(std::abs(pOut[i] - pIn_1[i] - pIn_2[i]) > epsilon)
		{
			printf("Wrong sum computation on CPU!\n");
			break;
		}
		if(std::abs(pOut_fromGPU[i] - pIn_1[i] - pIn_2[i]) > epsilon)
		{
			printf("Wrong sum computation on GPU!\n");
			break;
		}
	}

	if(i != N)
	{
		printf("Something's wrong... try again\n");	
	}
	else
	{
		printf("Success! Everythings ok\n");
	}

	// Libération de la mémoire à la fois pour le CPU et le GPU.
   free(pOut);      pOut    = NULL;
   free(pIn_1);     pIn_1   = NULL;
   free(pIn_2);     pIn_2   = NULL;
   free(pOut_fromGPU);     pOut_fromGPU   = NULL;
   HANDLE_ERROR(hipFree(dIn_1)); dIn_1   = NULL;
   HANDLE_ERROR(hipFree(dIn_2)); dIn_2   = NULL;
   HANDLE_ERROR(hipFree(dOut)); dOut   = NULL;
  }
  catch(...)
  {
	printf("Something's wrong... try again\n");	 
  }
}