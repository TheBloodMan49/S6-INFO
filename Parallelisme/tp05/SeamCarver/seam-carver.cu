#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <thread>
#include <string>
#include <iostream>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

// Handle error ///////////////////////////////////////////////////////////////

static void HandleError(	hipError_t err,
    const char *file,
    int line )
  {
    if (err != hipSuccess)
    {
      printf( "%s in %s at line %d\n", hipGetErrorString( err ),
      file, line );
      exit( EXIT_FAILURE );
    }
  }
  #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// ENERGY CPU /////////////////////////////////////////////////////////////////////

unsigned int energy_at_cpu(
    const unsigned char *data,
    int w,
    int h,
    int x,
    int y) {
int x0 = x == 0 ? x : x - 1;
int x1 = x == w - 1 ? x : x + 1;
int ix0 = (y * w + x0) * 3;
int ix1 = (y * w + x1) * 3;
unsigned int dxr = data[ix0    ] - data[ix1    ];
unsigned int dxg = data[ix0 + 1] - data[ix1 + 1];
unsigned int dxb = data[ix0 + 2] - data[ix1 + 2];
unsigned int dx = dxr * dxr + dxg * dxg + dxb * dxb;

int y0 = y == 0 ? y : y - 1;
int y1 = y == h - 1 ? y : y + 1;
int iy0 = (y0 * w + x) * 3;
int iy1 = (y1 * w + x) * 3;
unsigned int dyr = data[iy0    ] - data[iy1    ];
unsigned int dyg = data[iy0 + 1] - data[iy1 + 1];
unsigned int dyb = data[iy0 + 2] - data[iy1 + 2];
unsigned int dy = dyr * dyr + dyg * dyg + dyb * dyb;

return dx + dy;
}

unsigned int * compute_energy_cpu(const unsigned char *data, int w, int h) {
unsigned int *energy = (unsigned int *)malloc(w * h * sizeof(unsigned int));
if (!energy) {
    fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);
    return NULL;
}

for (int y = 0; y < h; y++)
{
    for (int x = 0; x < w; x++) 
    {
        int i = y * w + x;
        energy[i] = energy_at_cpu(data, w, h, x, y);
    }
}

return energy;
}

// ENERGY GPU /////////////////////////////////////////////////////////////////////

//compute energy with GPU version
__global__ void compute_energy_kernel(
    unsigned char * data_GPU,
    unsigned int * energy_GPU,
    int w,
    int h
)
{
    // Get x and y
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Dimension test
     if(x < w && y < h)
     {
         // Compute energy_GPU
       int x0 = x == 0 ? x : x - 1;
       int x1 = x == w - 1 ? x : x + 1;
       int ix0 = (y * w + x0) * 3;
       int ix1 = (y * w + x1) * 3;
       unsigned int dxr = data_GPU[ix0    ] - data_GPU[ix1    ];
       unsigned int dxg = data_GPU[ix0 + 1] - data_GPU[ix1 + 1];
       unsigned int dxb = data_GPU[ix0 + 2] - data_GPU[ix1 + 2];
       unsigned int dx = dxr * dxr + dxg * dxg + dxb * dxb;

       int y0 = y == 0 ? y : y - 1;
       int y1 = y == h - 1 ? y : y + 1;
       int iy0 = (y0 * w + x) * 3;
       int iy1 = (y1 * w + x) * 3;
       unsigned int dyr = data_GPU[iy0    ] - data_GPU[iy1    ];
       unsigned int dyg = data_GPU[iy0 + 1] - data_GPU[iy1 + 1];
       unsigned int dyb = data_GPU[iy0 + 2] - data_GPU[iy1 + 2];
       unsigned int dy = dyr * dyr + dyg * dyg + dyb * dyb;

       energy_GPU[y * w + x] = dx + dy;

     }
}

unsigned int * compute_energy(const unsigned char *data, int w, int h) {
    // grid and block size
    dim3 block_size(16,16);
    dim3 grid_size((w + block_size.x - 1) / block_size.x, (h + block_size.y - 1) / block_size.y);

    // Allocation
    int size = w * h;
    unsigned char * data_GPU = nullptr;
    unsigned int * energy_GPU = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&data_GPU, size * 3 * sizeof(unsigned char)));
    HANDLE_ERROR(hipMalloc((void**)&energy_GPU, size * sizeof(unsigned int)));

    // Fill data_GPU with data
    HANDLE_ERROR(hipMemcpy(data_GPU, data, size * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Call kernel
    compute_energy_kernel<<<grid_size, block_size>>>(data_GPU, energy_GPU, w, h);

    // Get energy map back to CPU
    unsigned int *energy = (unsigned int *)malloc(w * h * sizeof(unsigned int));
    if (!energy) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);
        return NULL;
    }
    // Fill energy with energy_GPU
    HANDLE_ERROR(hipMemcpy(energy, energy_GPU, size * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // Free GPU memory
    HANDLE_ERROR(hipFree(data_GPU));
    HANDLE_ERROR(hipFree(energy_GPU));
    
    return energy;
}

// SEAMS //////////////////////////////////////////////////////////////////////

struct seam_link {
    // The X and Y coordinates of the link are inferred by the position of the
    // link in a links array.

    // The minimal energy for any connected seam ending at this position.
    unsigned int energy;

    // The parent X coordinate for vertical seams, Y for horizontal seams.
    int parent_coordinate;
};

struct seam_link * compute_vertical_seam_links(
        const unsigned int *energy,
        int w,
        int h) {
    struct seam_link *links = (seam_link *)malloc(w * h * sizeof(struct seam_link));
    if (!links) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);
        return NULL;
    }

    for (int x = 0; x < w; x++) {
        links[x] = (struct seam_link) {
            .energy = energy[x],
            .parent_coordinate = -1
        };
    }

    for (int y = 1; y < h; y++)
    for (int x = 0; x < w; x++) {
        int i = y * w + x;

        int min_parent_energy = INT_MAX;
        int min_parent_x = -1;

        int parent_x = x == 0 ? x : x - 1;
        int parent_x_end = x == w - 1 ? x : x + 1;
        for (; parent_x <= parent_x_end; parent_x++) {
            int candidate_energy = links[(y - 1) * w + parent_x].energy;
            if (candidate_energy < min_parent_energy) {
                min_parent_energy = candidate_energy;
                min_parent_x = parent_x;
            }
        }

        links[i] = (struct seam_link) {
            .energy = energy[i] + min_parent_energy,
            .parent_coordinate = min_parent_x
        };
    }

    return links;
}

int * get_minimal_seam(
        const struct seam_link *seam_links,
        int num_seams,
        int seam_length) {
    int *minimal_seam = (int *)malloc(seam_length * sizeof(int));
    if (!minimal_seam) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);
    }

    int min_coordinate = -1;
    int min_energy = INT_MAX;

    for (int coordinate = 0; coordinate < num_seams; coordinate++) {
        int i = num_seams * (seam_length - 1) + coordinate;
        if (seam_links[i].energy < min_energy) {
            min_coordinate = coordinate;
            min_energy = seam_links[i].energy;
        }
    }

    int i = 0;
    int offset = min_coordinate;

    for (int d = 0; d < seam_length; d++) {
        minimal_seam[i++] = offset;

        struct seam_link end =
            seam_links[num_seams * (seam_length - 1 - d) + offset];

        offset = end.parent_coordinate;
    }

    return minimal_seam;
}

// REMOVAL ////////////////////////////////////////////////////////////////////

unsigned char * image_after_vertical_seam_removal(
        const unsigned char *original_data,
        const int *vertical_seam,
        int w,
        int h) {
    unsigned char *img = (unsigned char *)malloc((w - 1) * h * 3);
    if (!img) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);
        return NULL;
    }

    for (int y = 0; y < h; y++) {
        int seamx = vertical_seam[h - 1 - y];

        for (int x = 0, imgx = 0; imgx < w - 1; x++, imgx++) {
            if (x == seamx) { x++; }

            int    i = (y *  w      + x   ) * 3;
            int imgi = (y * (w - 1) + imgx) * 3;

            img[imgi    ] = original_data[i    ];
            img[imgi + 1] = original_data[i + 1];
            img[imgi + 2] = original_data[i + 2];
        }
    }

    return img;
}

// OUTPUT /////////////////////////////////////////////////////////////////////

int write_energy(
        const unsigned int *energy,
        int w,
        int h,
        const char *filename) {
    int result = 0;

    unsigned char *energy_normalized = (unsigned char *)malloc(w * h);
    if (!energy_normalized) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);

        result = 1;
        //goto cleanup;
    }

    int max_energy = 1;
    for (int y = 0; y < h; y++)
    for (int x = 0; x < w; x++) {
        int i = y * w + x;
        max_energy = energy[i] > max_energy ? energy[i] : max_energy;
    }

    for (int y = 0; y < h; y++)
    for (int x = 0; x < w; x++) {
        int i = y * w + x;
        energy_normalized[i] = (char) ((double) energy[i] / max_energy * 255);
    }

    printf("Writing to '%s'\n", filename);
    if (!stbi_write_jpg(filename, w, h, 1, energy_normalized, 80)) {
        fprintf(stderr, "Unable to write output (%d)\n", __LINE__);

        result = 1;
    }

    return result;
}

int draw_vertical_seam(
        const unsigned char *data,
        const int *minimal_vertical_seam,
        int w,
        int h,
        const char *filename) {
    int result = 0;

    unsigned char *data_with_seams = (unsigned char *)malloc(w * h * 3);
    if (!data_with_seams) {
        fprintf(stderr, "Unable to allocate memory (%d)\n", __LINE__);

        result = 1;
    }

    memcpy(data_with_seams, data, w * h * 3);

    for (int y = h - 1; y >= 0; y--) {
        int x = minimal_vertical_seam[h - 1 - y];
        int i = (y * w + x) * 3;

        data_with_seams[i    ] = 255;
        data_with_seams[i + 1] = 0;
        data_with_seams[i + 2] = 0;
    }

    printf("Writing to '%s'\n", filename);
    if (!stbi_write_jpg(filename, w, h, 3, data_with_seams, 80)) {
        fprintf(stderr, "Unable to write output (%d)\n", __LINE__);

        result = 1;
    }

    return result;
}

int draw_image(
        const unsigned char *data,
        int w,
        int h,
        const char *filename) {
    printf("Writing %dx%d image to '%s'\n", w, h, filename);
    return stbi_write_jpg(filename, w, h, 3, data, 80);
}

// MAIN ///////////////////////////////////////////////////////////////////////

void show_usage(const char *program) {
    fprintf(
            stderr,
            "USAGE:\n"
            "  %s <input-filename> <output-directory> <num-iterations> <gpu?>\n",
            program);
}

unsigned char * run_iteration(
        const char *output_directory,
        const unsigned char *data,
        int w,
        int h,
        int iteration,
        bool use_gpu) {

    unsigned int *energy = NULL;
    struct seam_link *vertical_seam_links = NULL;
    int *minimal_vertical_seam = NULL;
    unsigned char *output_data = NULL;

    char output_filename[1024];

    if (use_gpu) {
        energy = compute_energy(data, w, h);
    } else {
        energy = compute_energy_cpu(data, w, h);
    }

    if (!energy) { }

    if (iteration == 0) {
        snprintf(output_filename , 1024, "%s/img-energy%d.jpg", output_directory, iteration);
        if (write_energy(energy, w, h, output_filename)) {
        }
    }

    vertical_seam_links = compute_vertical_seam_links(energy, w, h);
    free(energy);
    if (!vertical_seam_links) { }

    minimal_vertical_seam = get_minimal_seam(vertical_seam_links, w, h);
    free(vertical_seam_links);

    snprintf(
            output_filename,
            1024,
            "%s/img-seam-%04d.jpg",
            output_directory,
            iteration);
    printf("Iteration: %d\n", iteration);
    // write output image 
    // if (draw_vertical_seam(
    //             data,
    //             minimal_vertical_seam,
    //             w,
    //             h,
    //             output_filename)) {
    // }

    output_data =
        image_after_vertical_seam_removal(data, minimal_vertical_seam, w, h);

    return output_data;
}

int main(int argc, char **argv) {
    if (argc != 5) {
        show_usage(argv[0]);
        return 1;
    }

    const char *input_filename = argv[1];
    const char *output_directory = argv[2];
    int num_iterations = atoi(argv[3]);

    bool use_gpu = strcmp(argv[4],"gpu") == 0;

    int result = 0;

    unsigned char *initial_img = NULL;
    unsigned char *data = NULL;

    printf("Reading '%s'\n", input_filename);

    int w, h, n;
    initial_img = stbi_load(input_filename, &w, &h, &n, 3);
    if (!initial_img) {
        fprintf(stderr, "Unable to read '%s'\n", input_filename);

        result = 1;
    }

    printf("Loaded %dx%d image\n", w, h);

    auto t0 = std::chrono::high_resolution_clock::now();
    data = initial_img;
    for (int i = 0; i < num_iterations; i++) {
        unsigned char *next_data =
            run_iteration(output_directory, data, w, h, i, use_gpu);

        if (!next_data) {
            fprintf(stderr, "Error running iteration %d\n", i);

            result = 1;
        }

        if (i > 0) { free(data); }
        data = next_data;
        w--;
    }
    auto t1 = std::chrono::high_resolution_clock::now();
    std::cout << "Seam carving image compute in " << std::chrono::duration<double>(t1-t0).count() << " seconds\n";    

    char resized_output_filename[1024];
    snprintf(resized_output_filename, 1024, "%s/img-%s.jpg", output_directory, use_gpu ? "gpu" : "cpu");
    if (!draw_image(data, w, h, resized_output_filename)) {
        fprintf(
                stderr,
                "\033[1;31mUnable to write %s\033[0m\n",
                resized_output_filename);
    }

    return result;
}
